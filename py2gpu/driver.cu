extern "C" {

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif


#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL2(call, success, failure) do { \
  hipError_t err = call; \
  if (err != hipSuccess) { \
    fprintf(stdout, "Cuda error in file '%s' in line %i : %s.\n", \
            __FILE__, __LINE__, hipGetErrorString(err)); \
    return failure; \
  } else return success; \
  } while(0);

#define CUDA_SAFE_CALL(call) CUDA_SAFE_CALL2(call, 0, -1)

#ifdef DEVICEEMU
EXPORT int emulating = 1;
#else
EXPORT int emulating = 0;
#endif

EXPORT void *drv_alloc(int size) {
  void *data;
  CUDA_SAFE_CALL2(hipMalloc(&data, size), data, NULL);
}

EXPORT int drv_free(void *data) {
  CUDA_SAFE_CALL(hipFree(data));
}

EXPORT int drv_htod(void *target, void *source, int size) {
  CUDA_SAFE_CALL(hipMemcpy(target, source, size, hipMemcpyHostToDevice));
}

EXPORT int drv_dtoh(void *target, void *source, int size) {
  CUDA_SAFE_CALL(hipMemcpy(target, source, size, hipMemcpyDeviceToHost));
}

}
